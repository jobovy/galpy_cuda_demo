#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"

// for cuda profiler
#include "hip/hip_runtime_api.h"

#define M_s 1.f // Solar mass
#define G 39.5f// Gravitational constant Solar mass, AU

// single precision CUDA function to be called on GPU
__device__ float potential_thingy(float x, float y) {
    return G * M_s * x / powf((powf(x, 2) + powf(y, 2)), 1.5f);
}

// euler method
__global__ void euler_integration(float *x_out, float *y_out, 
				  float *vx_out, float *vy_out, 
				  int n, int steps, int current_step, 
				  float dt, int n_intermediate_steps) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int ii;
    float tdt= dt/n_intermediate_steps;
    float tx,ty,tvx,tvy;
    tx= x_out[(n*current_step-n)+tid];
    ty= y_out[(n*current_step-n)+tid];
    tvx= vx_out[(n*current_step-n)+tid];
    tvy= vy_out[(n*current_step-n)+tid];
    while (tid < n){
      for (ii=0; ii < n_intermediate_steps; ii++) {
	// directly assigning to x_out[n*current_step+tid] etc. does not work
	// for some reason...
	tvx = tvx - potential_thingy(tx,ty)*tdt;
	tvy = tvy - potential_thingy(ty,tx)*tdt;
	tx = tx + tvx * tdt;
	ty = ty + tvy * tdt;
      }
      x_out[n*current_step+tid]= tx;
      y_out[n*current_step+tid]= ty;
      vx_out[n*current_step+tid]= tvx;
      vy_out[n*current_step+tid]= tvy;
      tid += gridDim.x * blockDim.x;
    }
}

// euler method for velocity component
__global__ void euler_integration_vx(float *x_out, float *y_out, float *vx_out, int n, int steps, int current_step, float dt) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n){
        vx_out[n*current_step+tid] = vx_out[(n*current_step-n)+tid] - potential_thingy(x_out[(n*current_step-n)+tid], y_out[(n*current_step-n)+tid]) * dt;
        tid += gridDim.x * blockDim.x;
    }
}

// euler method for position component
__global__ void euler_integration_x(float *x_out, float *vx_out, int n, int steps, int current_step, float dt) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n){
        x_out[n*current_step+tid] = x_out[(n*current_step-n)+tid] + vx_out[n*current_step+tid] * dt;
        tid += gridDim.x * blockDim.x;
    }
}

extern "C" int integrate_euler_cuda(float *x, float *y, float *vx, float *vy, float *x_out, float *y_out, float *vx_out,
                                    float *vy_out, int n, int steps, float dt,int n_intermediate_steps) {
    // dev_** variables for variables on CUDA device
    float *dev_x_out, *dev_y_out, *dev_vx_out, *dev_vy_out;

    // streams related constants and things
    const int nStreams = 1;

    // stream for kernel
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; ++i)
        hipStreamCreate(&stream[i]);

    // allocate the memory on the GPU (VRAM)
    // cudaMalloc docs: http://horacio9573.no-ip.org/cuda/group__CUDART__MEMORY_gc63ffd93e344b939d6399199d8b12fef.html
    hipMalloc((void**)&dev_x_out, steps * n * sizeof(float));
    hipMalloc((void**)&dev_y_out, steps * n * sizeof(float));
    hipMalloc((void**)&dev_vx_out, steps * n * sizeof(float));
    hipMalloc((void**)&dev_vy_out, steps * n * sizeof(float));

    // map the arrays x, y, vx, vy to the corresponding GPU array
    // cudaMemcpy docs: http://horacio9573.no-ip.org/cuda/group__CUDART__MEMORY_g48efa06b81cc031b2aa6fdc2e9930741.html
    hipMemcpy(&dev_x_out[0], &x[0], n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dev_y_out[0], &y[0], n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dev_vx_out[0], &vx[0], n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dev_vy_out[0], &vy[0], n * sizeof(float), hipMemcpyHostToDevice);

    // loop time, because time steps cannot be paralleled
    int cstep = 1;  // keep track of the time in integration
    while (cstep < steps){
        // integrate
      euler_integration<<<128, 128, 0, stream[0]>>>(dev_x_out, dev_y_out, dev_vx_out, dev_vy_out, n, steps, cstep, dt,n_intermediate_steps);
        hipMemcpyAsync(&vx_out[cstep*n], &dev_vx_out[cstep*n], n * sizeof(float), hipMemcpyDeviceToHost, stream[0]);
        hipMemcpyAsync(&vy_out[cstep*n], &dev_vy_out[cstep*n], n * sizeof(float), hipMemcpyDeviceToHost, stream[0]);
        hipMemcpyAsync(&x_out[cstep*n], &dev_x_out[cstep*n], n * sizeof(float), hipMemcpyDeviceToHost, stream[0]);
        hipMemcpyAsync(&y_out[cstep*n], &dev_y_out[cstep*n], n * sizeof(float), hipMemcpyDeviceToHost, stream[0]);
        // make sure above all finished to start next time step because next time step depends on this step
        hipDeviceSynchronize();
        cstep += 1;
    }

    // free the memory allocated on the GPU after integration, if really galpy, need to take care memory for real
    hipFree(dev_x_out);
    hipFree(dev_y_out);
    hipFree(dev_vx_out);
    hipFree(dev_vy_out);

    return 0;
}
